// file: spmm.cu.cc
//
// Using cusparse API to test SpMM performance.
//  author: guyue huang
//  date  : 2021/06/29
// compile: nvcc version >=11.0

#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // hipsparseSpMM (>= v11.0) or hipsparseScsrmm
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib> // std::rand(), RAND_MAX
#include <vector>
#include "spmm_util.hpp" // read_mtx

#define CUDA_CHECK(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CUSPARSE_CHECK(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

int main(int argc, const char **argv)
{

    /// check command-line argument

    if (argc < 2)
    {
        printf("Require command-line argument: name of the sparse matrix file in .mtx format.\n");
        return EXIT_FAILURE;
    }

    //
    // Load sparse matrix
    //

    int M;                               // number of A-rows
    int K;                               // number of A-columns
    int nnz;                             // number of non-zeros in A
    std::vector<int> csr_indptr_buffer;  // buffer for indptr array in CSR format
    std::vector<int> csr_indices_buffer; // buffer for indices (column-ids) array in CSR format
    // load sparse matrix from mtx file
    read_mtx_file(argv[1],
                  M,
                  K,
                  nnz,
                  csr_indptr_buffer,
                  csr_indices_buffer);
    printf("Finish reading matrix %d rows, %d columns, %d nnz. \nIgnore original values and use randomly generated values.\n", M, K, nnz);
    int N = 128; // number of B-columns
    if (argc == 3)
        N = atoi(argv[2]);
    // Create GPU arrays
    float *B_h = NULL, *C_h = NULL, *csr_values_h = NULL, *C_ref = NULL;
    float *B_d = NULL, *C_d = NULL, *csr_values_d = NULL;
    int *csr_indptr_d = NULL, *csr_indices_d = NULL;

    B_h = (float *)malloc(sizeof(float) * K * N);
    C_h = (float *)malloc(sizeof(float) * M * N);
    C_ref = (float *)malloc(sizeof(float) * M * N);
    csr_values_h = (float *)malloc(sizeof(float) * nnz);
    if (!B_h || !C_h || !C_ref || !csr_values_h)
    {
        printf("Host allocation failed.\n");
        return EXIT_FAILURE;
    }

    fill_random(csr_values_h, nnz);
    fill_random(B_h, K * N);

    CUDA_CHECK(hipMalloc((void **)&B_d, sizeof(float) * K * N));
    CUDA_CHECK(hipMalloc((void **)&C_d, sizeof(float) * M * N));
    CUDA_CHECK(hipMalloc((void **)&csr_values_d, sizeof(float) * nnz));
    CUDA_CHECK(hipMalloc((void **)&csr_indptr_d, sizeof(int) * (M + 1)));
    CUDA_CHECK(hipMalloc((void **)&csr_indices_d, sizeof(int) * nnz));

    CUDA_CHECK(hipMemcpy(B_d, B_h, sizeof(float) * K * N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(C_d, 0x0, sizeof(float) * M * N));
    CUDA_CHECK(hipMemcpy(csr_values_d, csr_values_h, sizeof(float) * nnz, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(csr_indptr_d, csr_indptr_buffer.data(), sizeof(int) * (M + 1), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(csr_indices_d, csr_indices_buffer.data(), sizeof(int) * nnz, hipMemcpyHostToDevice));

    //
    // Run SpMM and check result
    //

    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t csrDescr;
    hipsparseDnMatDescr_t dnMatInputDescr, dnMatOutputDescr;
    float alpha = 1.0f, beta = 0.0f;

    CUSPARSE_CHECK(hipsparseCreate(&handle));

    // creating sparse csr matrix
    CUSPARSE_CHECK(hipsparseCreateCsr(&csrDescr,
                                     M, K, nnz, csr_indptr_d, csr_indices_d, csr_values_d,
                                     HIPSPARSE_INDEX_32I, // index 32-integer for indptr
                                     HIPSPARSE_INDEX_32I, // index 32-integer for indices
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_32F // datatype: 32-bit float real number
                                     ));

    // creating dense matrices
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dnMatInputDescr,
                                       K,
                                       N,
                                       N,
                                       B_d,
                                       HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));
    CUSPARSE_CHECK(hipsparseCreateDnMat(&dnMatOutputDescr,
                                       M,
                                       N,
                                       N,
                                       C_d,
                                       HIP_R_32F,
                                       HIPSPARSE_ORDER_ROW));

    // allocate workspace buffer
    size_t workspace_size;
    CUSPARSE_CHECK(hipsparseSpMM_bufferSize(handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           csrDescr,
                                           dnMatInputDescr,
                                           &beta,
                                           dnMatOutputDescr,
                                           HIP_R_32F,
                                           HIPSPARSE_SPMM_ALG_DEFAULT,
                                           &workspace_size));

    void *workspace = NULL;
    CUDA_CHECK(hipMalloc(&workspace, workspace_size));

    // run SpMM
    CUSPARSE_CHECK(hipsparseSpMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                                HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                                &alpha,
                                csrDescr,
                                dnMatInputDescr,
                                &beta,
                                dnMatOutputDescr,
                                HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT,
                                workspace));

    CUDA_CHECK(hipMemcpy(C_h, C_d, sizeof(float) * M * N, hipMemcpyDeviceToHost));

    spmm_reference_host<int, float>(M, N, K, csr_indptr_buffer.data(), csr_indices_buffer.data(), csr_values_h, B_h, C_ref);

    bool correct = check_result<float>(M, N, C_h, C_ref);

    //
    // Benchmark SpMM performance
    //

    if (correct)
    {

        GpuTimer gpu_timer;
        int warmup_iter = 10;
        int repeat_iter = 100;
        for (int iter = 0; iter < warmup_iter + repeat_iter; iter++)
        {
            if (iter == warmup_iter)
            {
                gpu_timer.start();
            }

            hipsparseSpMM(handle,
                         HIPSPARSE_OPERATION_NON_TRANSPOSE, // opA
                         HIPSPARSE_OPERATION_NON_TRANSPOSE, // opB
                         &alpha,
                         csrDescr,
                         dnMatInputDescr,
                         &beta,
                         dnMatOutputDescr,
                         HIP_R_32F,
                         HIPSPARSE_SPMM_ALG_DEFAULT,
                         workspace);
        }
        gpu_timer.stop();

        float kernel_dur_msecs = gpu_timer.elapsed_msecs() / repeat_iter;

        float MFlop_count = (float)nnz / 1e6 * N * 2;

        float gflops = MFlop_count / kernel_dur_msecs;

        printf("Report: spmm A(%d x %d) * B(%d x %d) sparsity %f (nnz=%d) \n Time %f (ms), Throughput %f (gflops).\n",
               M, K, K, N,
               (float)nnz / M / K,
               nnz,
               kernel_dur_msecs,
               gflops);
    }

    /// free memory

    if (B_h)
        free(B_h);
    if (C_h)
        free(C_h);
    if (C_ref)
        free(C_ref);
    if (csr_values_h)
        free(csr_values_h);
    if (B_d)
        CUDA_CHECK(hipFree(B_d));
    if (C_d)
        CUDA_CHECK(hipFree(C_d));
    if (csr_values_d)
        CUDA_CHECK(hipFree(csr_values_d));
    if (csr_indptr_d)
        CUDA_CHECK(hipFree(csr_indptr_d));
    if (csr_indices_d)
        CUDA_CHECK(hipFree(csr_indices_d));
    if (workspace)
        CUDA_CHECK(hipFree(workspace));

    return EXIT_SUCCESS;
}
